#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */


#include "book.h"
#include "cpu_bitmap.h"

#define DIM 1024

struct hipComplex {
    float   r;
    float   i;
    __device__ hipComplex(float a, float b) : r(a), i(b) {}
    __device__ float magnitude2(void) {
        return r * r + i * i;
    }
    __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
    }
    __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r + a.r, i + a.i);
    }
};

__device__ int julia(int x, int y, float real, float img) {
    const float scale = 1.5;
    float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
    float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);

    //hipComplex c(-0.8, 0.156);
    hipComplex c(real, img);
    hipComplex a(jx, jy);

    int i = 0;
    for (i = 0; i < 200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}

__global__ void kernel(unsigned char* ptr) {
    // map from blockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // now calculate the value at that position
    int juliaValue_r = julia(x, y, -0.814, 0.152);
    int juliaValue_g = julia(x, y, -0.635, -0.123);
    int juliaValue_b = julia(x, y, -0.314, -0.618);
    ptr[offset * 4 + 0] = 255 * juliaValue_r;
    ptr[offset * 4 + 1] = 255 * juliaValue_g;
    ptr[offset * 4 + 2] = 255 * juliaValue_b;
    ptr[offset * 4 + 3] = 255;
}

// globals needed by the update routine
struct DataBlock {
    unsigned char* dev_bitmap;
};

int main(void) {
    DataBlock   data;
    CPUBitmap bitmap(DIM, DIM, &data);
    unsigned char* dev_bitmap;

    HANDLE_ERROR(hipMalloc((void**)&dev_bitmap, bitmap.image_size()));
    data.dev_bitmap = dev_bitmap;

    dim3    blocks(DIM / 16, DIM / 16);
    dim3    threads(16, 16);
    kernel << <blocks, threads >> > (dev_bitmap);

    HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap,
        bitmap.image_size(),
        hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(dev_bitmap));

    bitmap.display_and_exit();
}
